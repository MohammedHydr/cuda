#include "hip/hip_runtime.h"
#include "common.h"

#include "timer.h"

__global__ void rgb2gray_kernel (unsigned char* red, unsigned char* green, unsigned char* blue, unsigned char* gray, 
					unsigned int width, unsigned int height) {

    unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int i = row * width + col;
	
	if (row < height && col < width){
		gray[i] = red[i]*3/10 + green[i]*6/10 + blue[i]/10;
	}
}

void rgb2gray_gpu(unsigned char* red, unsigned char* green, unsigned char* blue, unsigned char* gray, 
					unsigned int width, unsigned int height) {

    Timer timer;

    // Allocate GPU memory
    startTime(&timer);
	unsigned char *red_d, *green_d, *blue_d, *gray_d;
	hipMalloc((void **) &red_d, height*width*sizeof(unsigned char));
	hipMalloc((void **) &green_d, height*width*sizeof(unsigned char));
	hipMalloc((void **) &blue_d, height*width*sizeof(unsigned char));
	hipMalloc((void **) &gray_d, height*width*sizeof(unsigned char));

    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Allocation time");

    // Copy data to GPU
    startTime(&timer);

    // TODO
	hipMemcpy(red_d, red, height*width*sizeof(unsigned char), hipMemcpyHostToDevice);
	hipMemcpy(green_d, green, height*width*sizeof(unsigned char), hipMemcpyHostToDevice);
	hipMemcpy(blue_d, blue, height*width*sizeof(unsigned char), hipMemcpyHostToDevice);


    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Copy to GPU time");

    // Call kernel
    startTime(&timer);

    // TODO					  x, y , z 	by default z is 1					
	dim3 numThreadsPerBlock(32, 32);
	dim3 numBlocks((width + numThreadsPerBlock.x - 1)/numThreadsPerBlock.x, (height + numThreadsPerBlock.y - 1)/numThreadsPerBlock.y);
	rgb2gray_kernel <<<numBlocks, numThreadsPerBlock>>>(red_d, green_d, blue_d, gray_d, width, height);


    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Kernel time", GREEN);

    // Copy data from GPU
    startTime(&timer);

    // TODO
	hipMemcpy(gray, gray_d, height*width*sizeof(unsigned char), hipMemcpyDeviceToHost);


    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Copy from GPU time");

    // Free GPU memory
    startTime(&timer);

    // TODO

	hipFree(red_d);
	hipFree(green_d);
	hipFree(blue_d);
	hipFree(gray_d);




    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Deallocation time");

}